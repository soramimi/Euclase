#include "hip/hip_runtime.h"
#include <stdio.h>
#include "libeuclasecuda.h"
#include <hip/hip_fp16.h>

#define API_FUNC_ENTRY(NAME) cuda_##NAME
#define GAMMA (2.2f)
#define gamma(X) powf(X, 1 / GAMMA)
#define degamma(X) powf(X, GAMMA)

__device__ inline uint8_t clamp_uint8(float x)
{
	return (uint8_t)max(0.0f, min(255.0f, x));
}

__device__ inline float clamp_f01(float x)
{
	return max(0.0f, min(1.0f, x));
}

__global__ void cu_round_brush(int w, int h, float cx, float cy, float radius, float blur, float mul, float *p)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < w && j < h) {
		float x = i + 0.5 - cx;
		float y = j + 0.5 - cy;

		float value = 0;
		float d = sqrtf(x * x + y * y);
		if (d > radius) {
			value = 0;
		} else if (d > blur && mul > 0) {
			float t = (d - blur) * mul;
			if (t < 1) {
				float u = 1 - t;
				value = u * u * (u + t * 3);
			}
		} else {
			value = 1;
		}

		p += 4 * (w * j + i);
		p[3] = value;
	}
}

void API_FUNC_ENTRY(round_brush)(int w, int h, float cx, float cy, float radius, float blur, float mul, cudamem_t *mem)
{
	dim3 blocks((w + 15) / 16, (h + 15) / 16);
	dim3 threads(16, 16);
	cu_round_brush<<<blocks,threads>>>(w, h, cx, cy, radius, blur, mul, (float *)mem);
}

__global__ void cu_saturation_brightness(int w, int h, int red, int green, int blue, uint8_t *p)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < w && y < h) {
		float t = x / (w - 1.0);
		float u = 1 - y / (h - 1.0);
		float r = (255 - (255 - red) * t) * u;
		float g = (255 - (255 - green) * t) * u;
		float b = (255 - (255 - blue) * t) * u;
		int z = 4 * (w * y + x);
		p[z + 0] = r;
		p[z + 1] = g;
		p[z + 2] = b;
		p[z + 3] = 255;
	}
}

void API_FUNC_ENTRY(saturation_brightness)(int w, int h, int red, int green, int blue, cudamem_t *mem)
{
	dim3 blocks((w + 15) / 16, (h + 15) / 16);
	dim3 threads(16, 16);
	cu_saturation_brightness<<<blocks,threads>>>(w, h, red, green, blue, (uint8_t *)mem);
}

__global__ void cu_fill_uint8_rgba_kernel(int w, int h, uint8_t r, uint8_t g, uint8_t b, uint8_t a, uint8_t *dst, int dst_w, int dx, int dy)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < w && y < h) {
		uint8_t *d = dst + 4 * (dst_w * (dy + y) + dx + x);
		d[0] = r;
		d[1] = g;
		d[2] = b;
		d[3] = a;
	}
}

void API_FUNC_ENTRY(fill_uint8_rgba)(int w, int h, uint8_t r, uint8_t g, uint8_t b, uint8_t a, cudamem_t *dst, int dst_w, int dst_h, int dx, int dy)
{
	uint8_t *d = (uint8_t *)dst;

	dim3 blocks((w + 15) / 16, (h + 15) / 16);
	dim3 threads(16, 16);
	cu_fill_uint8_rgba_kernel<<<blocks,threads>>>(w, h, r, g, b, a, d, dst_w, dx, dy);
}

__global__ void cu_fill_fp32_rgba_kernel(int w, int h, float r, float g, float b, float a, float *dst, int dst_w, int dx, int dy)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < w && y < h) {
		float *d = dst + 4 * (dst_w * (dy + y) + dx + x);
		d[0] = r;
		d[1] = g;
		d[2] = b;
		d[3] = a;
	}
}

void API_FUNC_ENTRY(fill_fp32_rgba)(int w, int h, float r, float g, float b, float a, cudamem_t *dst, int dst_w, int dst_h, int dx, int dy)
{
	float *d = (float *)dst;

	dim3 blocks((w + 15) / 16, (h + 15) / 16);
	dim3 threads(16, 16);
	cu_fill_fp32_rgba_kernel<<<blocks,threads>>>(w, h, r, g, b, a, d, dst_w, dx, dy);
}

__global__ void cu_fill_fp16_rgba_kernel(int w, int h, float r, float g, float b, float a, __half *dst, int dst_w, int dx, int dy)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < w && y < h) {
		__half *d = dst + 4 * (dst_w * (dy + y) + dx + x);
		d[0] = r;
		d[1] = g;
		d[2] = b;
		d[3] = a;
	}
}

void API_FUNC_ENTRY(fill_fp16_rgba)(int w, int h, float r, float g, float b, float a, cudamem_t *dst, int dst_w, int dst_h, int dx, int dy)
{
	__half *d = (__half *)dst;

	dim3 blocks((w + 15) / 16, (h + 15) / 16);
	dim3 threads(16, 16);
	cu_fill_fp16_rgba_kernel<<<blocks,threads>>>(w, h, r, g, b, a, d, dst_w, dx, dy);
}

void API_FUNC_ENTRY(copy_uint8_rgba)(int w, int h, cudamem_t const *src, int src_w, int src_h, int sx, int sy, cudamem_t *dst, int dst_w, int dst_h, int dx, int dy)
{
	uint32_t const *s = (uint32_t const *)src;
	uint32_t *d = (uint32_t *)dst;

	if (w == src_w && w == dst_w && sx == 0 && sy == 0 && dx == 0 && dy == 0) {
		hipMemcpy(d, s, 4 * w * h, hipMemcpyDeviceToHost);
	} else {
		s += src_w * sy + sx;
		d += dst_w * dy + dx;
		for (int y = 0; y < h; y++) {
			hipMemcpy(d + dx, s + sx, 4 * w, hipMemcpyDeviceToHost);
			s += src_w;
			d += dst_w;
		}
	}
}

__device__ void alpha_blend_fp32_RGBA(float *d, float const *s, float m)
{
	float baseR = d[0];
	float baseG = d[1];
	float baseB = d[2];
	float baseA = d[3];
	float overR = s[0];
	float overG = s[1];
	float overB = s[2];
	float overA = s[3];
	overA = overA * m;
	float r = overR * overA + baseR * baseA * (1 - overA);
	float g = overG * overA + baseG * baseA * (1 - overA);
	float b = overB * overA + baseB * baseA * (1 - overA);
	float a = overA + baseA * (1 - overA);
	if (a > 0) {
		float t = 1 / a;
		r *= t;
		g *= t;
		b *= t;
	}
	d[0] = r;
	d[1] = g;
	d[2] = b;
	d[3] = a;
}

__device__ void alpha_blend_fp16_RGBA(__half *d, __half const *s, __half m)
{
	__half baseR = d[0];
	__half baseG = d[1];
	__half baseB = d[2];
	__half baseA = d[3];
	__half overR = s[0];
	__half overG = s[1];
	__half overB = s[2];
	__half overA = s[3];
	overA = overA * m;
	__half r = overR * overA + baseR * baseA * ((__half)1 - overA);
	__half g = overG * overA + baseG * baseA * ((__half)1 - overA);
	__half b = overB * overA + baseB * baseA * ((__half)1 - overA);
	__half a = overA + baseA * ((__half)1 - overA);
	if (a > (__half)0) {
		__half t = (__half)1 / a;
		r *= t;
		g *= t;
		b *= t;
	}
	d[0] = r;
	d[1] = g;
	d[2] = b;
	d[3] = a;
}

__device__ void alpha_blend_float_GrayA(float *d, float const *s, float m)
{
	float baseV = d[0];
	float baseA = d[1];
	float overV = s[0];
	float overA = s[1];
	overA = overA * m;
	float r = overV * overA + baseV * baseA * (1 - overA);
	float a = overA + baseA * (1 - overA);
	if (a > 0) {
		r /= a;
	}
	d[0] = r;
	d[1] = a;
}

__global__ void cu_blend_fp32_RGBA_kernel(int w, int h, float const *src, int src_w, int sx, int sy, uint8_t const *mask, int mask_w, float *dst, int dst_w, int dx, int dy)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < w && y < h) {
		float const *s = src + 4 * (src_w * (sy + y) + sx + x);
		float *d = dst + 4 * (dst_w * (dy + y) + dx + x);
		float m = mask ? mask[mask_w * y + x] / 255.0f : 1.0f;
		alpha_blend_fp32_RGBA(d, s, m);
	}
}

__global__ void cu_blend_fp16_RGBA_kernel(int w, int h, __half const *src, int src_w, int sx, int sy, uint8_t const *mask, int mask_w, __half *dst, int dst_w, int dx, int dy)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < w && y < h) {
		__half const *s = src + 4 * (src_w * (sy + y) + sx + x);
		__half *d = dst + 4 * (dst_w * (dy + y) + dx + x);
		__half m = mask ? mask[mask_w * y + x] / 255.0f : 1.0f;
		alpha_blend_fp16_RGBA(d, s, m);
	}
}

void API_FUNC_ENTRY(blend_fp32_RGBA)(int w, int h, cudamem_t const *src, int src_w, int src_h, int sx, int sy, cudamem_t const *mask, int mask_w, int mask_h, cudamem_t *dst, int dst_w, int dst_h, int dx, int dy)
{
	float const *s = (float const *)src;
	float *d = (float *)dst;
	uint8_t *buf_mask = nullptr;

	if (mask) {
		hipMalloc(&buf_mask, sizeof(uint8_t) * mask_w * mask_h);
		hipMemcpy(buf_mask, mask, sizeof(uint8_t) * mask_w * mask_h, hipMemcpyHostToDevice);
	}

	dim3 blocks((w + 15) / 16, (h + 15) / 16);
	dim3 threads(16, 16);
	cu_blend_fp32_RGBA_kernel<<<blocks,threads>>>(w, h, s, src_w, sx, sy, buf_mask, mask_w, d, dst_w, dx, dy);

	if (mask) {
		hipFree(buf_mask);
	}
}

void API_FUNC_ENTRY(blend_fp16_RGBA)(int w, int h, cudamem_t const *src, int src_w, int src_h, int sx, int sy, cudamem_t const *mask, int mask_w, int mask_h, cudamem_t *dst, int dst_w, int dst_h, int dx, int dy)
{
	__half const *s = (__half const *)src;
	__half *d = (__half *)dst;
	uint8_t *buf_mask = nullptr;

	if (mask) {
		hipMalloc(&buf_mask, sizeof(uint8_t) * mask_w * mask_h);
		hipMemcpy(buf_mask, mask, sizeof(uint8_t) * mask_w * mask_h, hipMemcpyHostToDevice);
	}

	dim3 blocks((w + 15) / 16, (h + 15) / 16);
	dim3 threads(16, 16);
	cu_blend_fp16_RGBA_kernel<<<blocks,threads>>>(w, h, s, src_w, sx, sy, buf_mask, mask_w, d, dst_w, dx, dy);

	if (mask) {
		hipFree(buf_mask);
	}
}

__global__ void cu_erase_fp32_RGBA_kernel(int w, int h, float const *src, int src_w, int sx, int sy, uint8_t const *mask, int mask_w, float *dst, int dst_w, int dx, int dy)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < w && y < h) {
		float const *s = src + 4 * (src_w * (sy + y) + sx + x);
		float *d = dst + 4 * (dst_w * (dy + y) + dx + x);
		float m = mask ? mask[mask_w * y + x] / 255.0f : 1.0f;
		d[3] *= 1.0f - clamp_f01(s[3] * m);
	}
}

__global__ void cu_erase_fp16_RGBA_kernel(int w, int h, __half const *src, int src_w, int sx, int sy, uint8_t const *mask, int mask_w, __half *dst, int dst_w, int dx, int dy)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < w && y < h) {
		__half const *s = src + 4 * (src_w * (sy + y) + sx + x);
		__half *d = dst + 4 * (dst_w * (dy + y) + dx + x);
		__half m = mask ? mask[mask_w * y + x] / 255.0f : 1.0f;
		d[3] *= 1.0f - clamp_f01(s[3] * m);
	}
}

void API_FUNC_ENTRY(erase_fp32_RGBA)(int w, int h, cudamem_t const *src, int src_w, int src_h, int sx, int sy, cudamem_t const *mask, int mask_w, int mask_h, cudamem_t *dst, int dst_w, int dst_h, int dx, int dy)
{
	float const *s = (float const *)src;
	float *d = (float *)dst;
	uint8_t *buf_mask = nullptr;

	if (mask) {
		hipMalloc(&buf_mask, sizeof(uint8_t) * mask_w * mask_h);
		hipMemcpy(buf_mask, mask, sizeof(uint8_t) * mask_w * mask_h, hipMemcpyHostToDevice);
	}

	dim3 blocks((w + 15) / 16, (h + 15) / 16);
	dim3 threads(16, 16);
	cu_erase_fp32_RGBA_kernel<<<blocks,threads>>>(w, h, s, src_w, sx, sy, buf_mask, mask_w, d, dst_w, dx, dy);

	if (mask) {
		hipFree(buf_mask);
	}
}

void API_FUNC_ENTRY(erase_fp16_RGBA)(int w, int h, cudamem_t const *src, int src_w, int src_h, int sx, int sy, cudamem_t const *mask, int mask_w, int mask_h, cudamem_t *dst, int dst_w, int dst_h, int dx, int dy)
{
	__half const *s = (__half const *)src;
	__half *d = (__half *)dst;
	uint8_t *buf_mask = nullptr;

	if (mask) {
		hipMalloc(&buf_mask, sizeof(uint8_t) * mask_w * mask_h);
		hipMemcpy(buf_mask, mask, sizeof(uint8_t) * mask_w * mask_h, hipMemcpyHostToDevice);
	}

	dim3 blocks((w + 15) / 16, (h + 15) / 16);
	dim3 threads(16, 16);
	cu_erase_fp16_RGBA_kernel<<<blocks,threads>>>(w, h, s, src_w, sx, sy, buf_mask, mask_w, d, dst_w, dx, dy);

	if (mask) {
		hipFree(buf_mask);
	}
}

__global__ void cu_blend_uint8_grayscale_kernel(int w, int h, uint8_t const *src, int src_w, int sx, int sy, uint8_t const *mask, int mask_w, uint8_t *dst, int dst_w, int dx, int dy)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < w && y < h) {
		uint8_t m = mask ? *(mask + mask_w * y + x) : 255;
		uint8_t const *s = src + (src_w * (sy + y) + sx + x);
		uint8_t *d = dst + (dst_w * (dy + y) + dx + x);
		*d = (*d * (255 - m) + *s * m) / 255;
	}
}

void API_FUNC_ENTRY(blend_uint8_grayscale)(int w, int h, cudamem_t const *src, int src_w, int src_h, int sx, int sy, cudamem_t const *mask, int mask_w, int mask_h, cudamem_t *dst, int dst_w, int dst_h, int dx, int dy)
{
	uint8_t const *s = (uint8_t const *)src;
	uint8_t *d = (uint8_t *)dst;
	uint8_t *buf_mask = nullptr;

	if (mask) {
		hipMalloc(&buf_mask, sizeof(uint8_t) * mask_w * mask_h);
		hipMemcpy(buf_mask, mask, sizeof(uint8_t) * mask_w * mask_h, hipMemcpyHostToDevice);
	}

	dim3 blocks((w + 15) / 16, (h + 15) / 16);
	dim3 threads(16, 16);
	cu_blend_uint8_grayscale_kernel<<<blocks,threads>>>(w, h, s, src_w, sx, sy, buf_mask, mask_w, d, dst_w, dx, dy);

	if (mask) {
		hipFree(buf_mask);
	}
}

__global__ void cu_outline_uint8_grayscale_kernel(int w, int h, uint8_t const *src, uint8_t *dst)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < w && y < h) {
		uint8_t *d = dst + w * y + x;
		if (x > 0 && x + 1 < w && y > 0 && y + 1 < h) {
			uint8_t const *s0 = src + w * (y - 1);
			uint8_t const *s1 = src + w * y;
			uint8_t const *s2 = src + w * (y + 1);
			uint8_t v = ~(s0[x - 1] & s0[x] & s0[x + 1] & s1[x - 1] & s1[x + 1] & s2[x - 1] & s2[x] & s2[x + 1]) & s1[x];
			*d = (v & 0x80) ? 0 : 255;
		} else {
			*d = 255;
		}
	}
}

void API_FUNC_ENTRY(outline_uint8_grayscale)(int w, int h, cudamem_t const *src, cudamem_t *dst)
{
	uint8_t const *s = (uint8_t const *)src;
	uint8_t *d = (uint8_t *)dst;

	dim3 blocks((w + 15) / 16, (h + 15) / 16);
	dim3 threads(16, 16);
	cu_outline_uint8_grayscale_kernel<<<blocks,threads>>>(w, h, s, d);
}

void API_FUNC_ENTRY(compose_fp32_rgba)(int w, int h, cudamem_t *dst, cudamem_t const *src, cudamem_t const *mask)
{
	cuda_blend_fp32_RGBA(w, h, src, w, h, 0, 0, mask, w, h, dst, w, h, 0, 0);
}

void API_FUNC_ENTRY(compose_fp16_rgba)(int w, int h, cudamem_t *dst, cudamem_t const *src, cudamem_t const *mask)
{
	cuda_blend_fp16_RGBA(w, h, src, w, h, 0, 0, mask, w, h, dst, w, h, 0, 0);
}

__global__ void cu_scale_fp32_to_uint8_rgba_kernel(int dw, int dh, int dstride, uint8_t *dst, int sw, int sh, float const *src)
{
	int dx = blockIdx.x * blockDim.x + threadIdx.x;
	int dy = blockIdx.y * blockDim.y + threadIdx.y;

	if (dx < dw && dy < dh) {
		int sx = dx * sw / dw;
		int sy = dy * sh / dh;
		float const *s = src + 4 * (sw * sy + sx);
		uint8_t *d = dst + 4 * (dstride * dy + dx);
		float R = max(0.0f, min(1.0f, s[0]));
		float G = max(0.0f, min(1.0f, s[1]));
		float B = max(0.0f, min(1.0f, s[2]));
		float A = max(0.0f, min(1.0f, s[3]));
		d[0] = int(gamma(R) * 255 + 0.5f);
		d[1] = int(gamma(G) * 255 + 0.5f);
		d[2] = int(gamma(B) * 255 + 0.5f);
		d[3] = int(A * 255 + 0.5f);
	}
}

__global__ void cu_scale_fp16_to_uint8_rgba_kernel(int dw, int dh, int dstride, uint8_t *dst, int sw, int sh, __half const *src)
{
	int dx = blockIdx.x * blockDim.x + threadIdx.x;
	int dy = blockIdx.y * blockDim.y + threadIdx.y;

	if (dx < dw && dy < dh) {
		int sx = dx * sw / dw;
		int sy = dy * sh / dh;
		__half const *s = src + 4 * (sw * sy + sx);
		uint8_t *d = dst + 4 * (dstride * dy + dx);
		float R = max(0.0f, min(1.0f, s[0]));
		float G = max(0.0f, min(1.0f, s[1]));
		float B = max(0.0f, min(1.0f, s[2]));
		float A = max(0.0f, min(1.0f, s[3]));
		d[0] = int(gamma(R) * 255 + 0.5f);
		d[1] = int(gamma(G) * 255 + 0.5f);
		d[2] = int(gamma(B) * 255 + 0.5f);
		d[3] = int(A * 255 + 0.5f);
	}
}

void API_FUNC_ENTRY(scale_fp32_to_uint8_rgba)(int dw, int dh, int dstride, cudamem_t *dst, int sw, int sh, cudamem_t const *src)
{
	dim3 blocks((dw + 15) / 16, (dh + 15) / 16);
	dim3 threads(16, 16);
	cu_scale_fp32_to_uint8_rgba_kernel<<<blocks,threads>>>(dw, dh, dstride, (uint8_t *)dst, sw, sh, (float const *)src);
}

void API_FUNC_ENTRY(scale_fp16_to_uint8_rgba)(int dw, int dh, int dstride, cudamem_t *dst, int sw, int sh, cudamem_t const *src)
{
	dim3 blocks((dw + 15) / 16, (dh + 15) / 16);
	dim3 threads(16, 16);
	cu_scale_fp16_to_uint8_rgba_kernel<<<blocks,threads>>>(dw, dh, dstride, (uint8_t *)dst, sw, sh, (__half const *)src);
}

__global__ void cu_scale_kernel(int dw, int dh, int dstride, uint8_t *dst, int sw, int sh, int sstride, uint8_t const *src, int psize)
{
	int dx = blockIdx.x * blockDim.x + threadIdx.x;
	int dy = blockIdx.y * blockDim.y + threadIdx.y;

	if (dx < dw && dy < dh) {
		int sx = dx * sw / dw;
		int sy = dy * sh / dh;
		uint8_t const *s = src + sstride * sy + sx * psize;
		uint8_t *d = dst + dstride * dy + dx * psize;
		memcpy(d, s, psize);
	}
}

void API_FUNC_ENTRY(scale)(int dw, int dh, int dstride, cudamem_t *dst, int sw, int sh, int sstride, cudamem_t const *src, int psize)
{
	dim3 blocks((dw + 15) / 16, (dh + 15) / 16);
	dim3 threads(16, 16);
	cu_scale_kernel<<<blocks,threads>>>(dw, dh, dstride, (uint8_t *)dst, sw, sh, sstride, (uint8_t const *)src, psize);
}

cudamem_t *API_FUNC_ENTRY(malloc)(int len)
{
	cudamem_t *mem = nullptr;
	hipMalloc((void **)&mem, len);
	return mem;
}

void API_FUNC_ENTRY(free)(cudamem_t *mem)
{
	hipFree(mem);
}

void API_FUNC_ENTRY(memcpy_htoh)(void *dst_h, void const *src_h, int len)
{
	hipMemcpy(dst_h, src_h, len, hipMemcpyHostToHost);
}

void API_FUNC_ENTRY(memcpy_dtoh)(void *dst_h, cudamem_t const *src_d, int len)
{
	hipMemcpy(dst_h, src_d, len, hipMemcpyDeviceToHost);
}

void API_FUNC_ENTRY(memcpy_htod)(cudamem_t *dst_d, void const *src_h, int len)
{
	hipMemcpy(dst_d, src_h, len, hipMemcpyHostToDevice);
}

void API_FUNC_ENTRY(memcpy_dtod)(cudamem_t *dst_d, cudamem_t const *src_d, int len)
{
	hipMemcpy(dst_d, src_d, len, hipMemcpyDeviceToDevice);
}

void API_FUNC_ENTRY(memset)(cudamem_t *dst, uint8_t c, int len)
{
	hipMemset(dst, c, len);
}

__global__ void cu_init_cudaplugin(uint8_t *p)
{
	int i = blockIdx.x;
	int j = threadIdx.x;
	p[i * 9 + j] = (i + 1) * (j + 1);
}

CUDAIMAGE_API api;

#ifdef _WIN32
extern "C" __declspec(dllexport) CUDAIMAGE_API const *init_cudaplugin(int n)
#else
extern "C" CUDAIMAGE_API const *init_cudaplugin(int n)
#endif
{
	if (n != sizeof(CUDAIMAGE_API)) return nullptr;

	uint8_t table[81];
	uint8_t *mem;
	hipMalloc((cudamem_t **)&mem, 81);
	dim3 b(9);
	dim3 t(9);
	cu_init_cudaplugin<<<b,t>>>(mem);
	hipMemcpy(table, mem, 81, hipMemcpyDeviceToHost);
	hipFree(mem);

	for (int i = 0; i < 9; i++) {
		for (int j = 0; j < 9; j++) {
			if (table[9 * i + j] != (i + 1) * (j + 1)) {
				return nullptr;
			}
		}
	}

#define API_FUNC(NAME) api.NAME = cuda_##NAME

	API_FUNC(malloc);
	API_FUNC(free);
	API_FUNC(memcpy_htoh);
	API_FUNC(memcpy_dtoh);
	API_FUNC(memcpy_htod);
	API_FUNC(memcpy_dtod);
	API_FUNC(memset);
	API_FUNC(saturation_brightness);
	API_FUNC(round_brush);
	API_FUNC(fill_uint8_rgba);
	API_FUNC(fill_fp32_rgba);
	API_FUNC(fill_fp16_rgba);
	API_FUNC(copy_uint8_rgba);
	API_FUNC(blend_fp32_RGBA);
	API_FUNC(blend_fp16_RGBA);
	API_FUNC(erase_fp32_RGBA);
	API_FUNC(erase_fp16_RGBA);
	API_FUNC(blend_uint8_grayscale);
	API_FUNC(outline_uint8_grayscale);
	API_FUNC(compose_fp32_rgba);
	API_FUNC(compose_fp16_rgba);
	API_FUNC(scale_fp32_to_uint8_rgba);
	API_FUNC(scale_fp16_to_uint8_rgba);
	API_FUNC(scale);

	return &api;
}
